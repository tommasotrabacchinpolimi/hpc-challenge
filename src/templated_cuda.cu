
#include <iostream>
#include <hip/hip_runtime.h>

#include <chrono>
#define GRID_SIZE 1000
#define BLOCK_SIZE 512
#define WARP_SIZE 32


void check_cuda(const std::string& msg) {
    hipDeviceSynchronize();
    hipError_t err;
    err = hipGetLastError();
    if(err != hipSuccess) {
        std::cout << "cuda error: " << msg << std::endl;
        std::cout << "description: " << err << std::endl;
    }
}

double dot(const double * x, const double * y, size_t size)
{
    double result = 0.0;
    for(size_t i = 0; i < size; i++)
    {
        result += x[i] * y[i];
    }
    return result;
}



void axpby(double alpha, const double * x, double beta, double * y, size_t size)
{
    // y = alpha * x + beta * y

    for(size_t i = 0; i < size; i++)
    {
        y[i] = alpha * x[i] + beta * y[i];
    }
}



void gemv(double alpha, const double * A, const double * x, double beta, double * y, size_t num_rows, size_t num_cols)
{
    // y = alpha * A * x + beta * y;

    for(size_t r = 0; r < num_rows; r++)
    {
        double y_val = 0.0;
        for(size_t c = 0; c < num_cols; c++)
        {
            y_val += alpha * A[r * num_cols + c] * x[c];
        }
        y[r] = beta * y[r] + y_val;
    }
}

void generate_matrix(size_t n, double** matrix_out) {
    auto* matrix = new double[n * n];
    for(size_t i = 0; i < n * n; i++) {
        matrix[i] = 0.0;
    }
    for(size_t i = 0; i < n; i++) {
        matrix[i*n + i] = 2.0;
        if(i != n-1) {
            matrix[(i+1)*n + i] = -1;
            matrix[i*n + (i+1)] = -1;
        }
    }
    *matrix_out = matrix;
}

void generate_rhs(size_t n, double value, double** rhs_out) {
    auto* rhs = new double[n];
    for(size_t i = 0; i < n; i++) {
        rhs[i] = value;
    }
    *rhs_out = rhs;
}


template<int blockSize>
__device__ void reduce_ws(double* __restrict__ data, float* __restrict__ out) {
    __shared__ float sdata[WARP_SIZE];
    int tid = threadIdx.x;
    float val;
    unsigned mask = 0xFFFFFFFFU;
    int lane = threadIdx.x % WARP_SIZE;
    int warpID = threadIdx.x / WARP_SIZE;
    val = data[tid];
    for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(mask, val, offset);
    }
    if (lane == 0){
        sdata[warpID] = val;
    }
    __syncthreads();

    if (warpID == 0){
        val = (tid < blockSize/WARP_SIZE)?sdata[lane]:0;
        for (int offset = WARP_SIZE/2; offset > 0; offset >>= 1) {
            val += __shfl_down_sync(mask, val, offset);
        }

        if (tid == 0) {
            atomicAdd(out, val);
        }
    }
}



template<int blockSize>
__device__ void warpReduce(volatile double* sdata, int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template<int blockSize>
__device__ void reduce(double* sdata, int tid) {
    if (blockSize >= 1024) {
        if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >= 512) {
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) {
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) {
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
}


template<int blockSize>
__device__ void row_column_mult_ws(const double* __restrict__ A, unsigned int row, int size, const double* __restrict__ p, double* __restrict__ Ap) {
    __shared__ double sArr[blockSize];
    __shared__ float partial;
    if(threadIdx.x == 0) {
        partial = 0.0;
    }
    for(unsigned int i = threadIdx.x; i < size + threadIdx.x; i+=2*blockSize) {
        //sArr[threadIdx.x] = ((i<size)?A[row*size + i]*p[i]:0.0) + ((i + blockSize<size)?A[row*size + i + blockSize]*(p[i + blockSize]):0.0);

        if(i < size && i + blockSize < size) {
            sArr[threadIdx.x] = fma(A[row*size + i],p[i], A[row*size + i + blockSize] * p[i + blockSize]);
        } else if(i < size){
            sArr[threadIdx.x] = A[row*size + i]*p[i];
        } else {
            sArr[threadIdx.x] = 0.0;
        }

        __syncthreads();
        reduce_ws<blockSize>(sArr, &partial);
    }
    if(threadIdx.x == 0) {
        Ap[row] = partial;
    }

}


template<int blockSize>
__device__ void row_column_mult(const double* __restrict__ A, unsigned int row, int size, const double* __restrict__ p, double* __restrict__ Ap) {
    __shared__ double sArr[blockSize];
    __shared__ double partial;
    if(threadIdx.x == 0) {
        partial = 0.0;
    }
    for(unsigned int i = threadIdx.x; i < size + threadIdx.x; i+=2*blockSize) {
        sArr[threadIdx.x] = ((i<size)?A[row*size + i]*p[i]:0.0) + ((i + blockSize<size)?A[row*size + i + blockSize]*(p[i + blockSize]):0.0);
        __syncthreads();
        reduce<blockSize>(sArr, threadIdx.x);
        if(threadIdx.x == 0) {
            partial += sArr[0];
        }
    }
    if(threadIdx.x == 0) {
        Ap[row] = partial;
    }

}

template<int blockSize>
__global__ void tiled_matrix_vector_mult(const double* __restrict__ A, const double* __restrict__ p, double* __restrict__ Ap, const unsigned int size) {
    __shared__ double sArr[blockSize];
    double Ap_partial = 0;
    const int tid = threadIdx.x + blockSize * blockIdx.x;
    for(unsigned int k = 0; k < (size - 1 + blockSize)/blockSize; k++) {
        sArr[threadIdx.x] = (k*blockSize + threadIdx.x < size) ? p[k*blockSize + threadIdx.x] : 0.0;
        __syncthreads();
        for(unsigned int e = 0; e < blockSize; e++) {
            Ap_partial += (tid + size * (k*blockSize + e) < size*size)?(A[tid + size * (k*blockSize + e)] * sArr[e]):0.0;
        }
        __syncthreads();
    }
    if(tid < size) {
        Ap[tid] = Ap_partial;
    }
}


template<int gridSize, int blockSize>
__global__ void matrix_vector_kernel(const double* __restrict__ A, double* __restrict__ p, double* __restrict__ Ap, int size) {
    for(unsigned int i = blockIdx.x; i < size; i+=gridSize) {
        //row_column_mult<blockSize>(A,i,size,p,Ap);
        row_column_mult_ws<blockSize>(A,i,size,p,Ap);
    }

}

template<int gridSize, int blockSize>
void matrix_vector_mult(const double* __restrict__ A, double* __restrict__ p, double* __restrict__ Ap, int size, hipStream_t stream) {
    //tiled_matrix_vector_mult<blockSize><<<(size  + blockSize)/blockSize, blockSize>>>(A, p, Ap, size);
    matrix_vector_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(A, p, Ap, size);
    //matrix_vector_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(A, p, Ap, size);
}


template<int blockSize>
__global__ void sumArray(const double* __restrict__ array, int size, double* __restrict__ result) {
    __shared__ double sArr[blockSize];
    __shared__ double partial;
    if(threadIdx.x == 0) {
        partial = 0;
    }
    for(unsigned int i = threadIdx.x; i < size + threadIdx.x; i+=2*blockSize) {
        sArr[threadIdx.x] = ((i<size)?array[i]:0.0) + ((i + blockSize < size)?array[i + blockSize]:0.0);
        __syncthreads();
        reduce<blockSize>(sArr, threadIdx.x);
        if(threadIdx.x == 0) {
            partial += sArr[0];
        }

    }
    if(threadIdx.x == 0) {
        *result = partial;
    }
}

template<int gridSize, int blockSize>
__global__ void dot_product_kernel(const double* __restrict__ x, const double* __restrict__ y, double* __restrict__ outArray, int size) {
    __shared__ double sArr[blockSize];
    if(threadIdx.x == 0) {
        outArray[blockIdx.x] = 0.0;
    }
    for(unsigned int i = blockIdx.x; 2*blockSize*i < size; i+=gridSize) {
        int tmp = i*2*blockSize + threadIdx.x;
        sArr[threadIdx.x] = ((tmp<size)?x[tmp]*y[tmp]:0.0) + ((tmp + blockSize<size)?x[tmp + blockSize]*y[tmp + blockSize]:0.0);
        __syncthreads();
        reduce<blockSize>(sArr, threadIdx.x);
        if(threadIdx.x == 0) {
            outArray[blockIdx.x] += sArr[0];
        }
    }
}

template<int gridSize, int blockSize>
void dot_product(const double* __restrict__ x, const double* __restrict__ y, double* __restrict__ outArray, int size, double* __restrict__ result, hipStream_t stream) {
    dot_product_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(x, y, outArray, size);
    sumArray<blockSize><<<1, blockSize>>>(outArray, gridSize, result);
}

template<int gridSize, int blockSize>
__global__ void axpby_kernel(const double* __restrict__ alpha, const double* __restrict__ x, double* __restrict__ y, int size) {
    int th_id = threadIdx.x + blockIdx.x * blockSize;
    if(th_id < size) {
        y[th_id] = (*alpha) * x[th_id] + y[th_id];
    }
}


template<int gridSize, int blockSize>
__global__ void _minus_axpby_kernel(const double* __restrict__ alpha, const double* __restrict__ x, double* __restrict__ y, int size) {
    int th_id = threadIdx.x + blockIdx.x * blockSize;
    if(th_id < size) {
        y[th_id] = -(*alpha) * x[th_id] + y[th_id];
    }
}

template<int gridSize, int blockSize>
__global__ void xpby_kernel( const double* __restrict__ x, double* __restrict__ y, const double* __restrict__ beta, int size) {
    int th_id = threadIdx.x + blockIdx.x * blockSize;
    if(th_id < size) {
        y[th_id] = (x[th_id] + (*beta) * y[th_id]);
    }
}



__global__ void divide(const double* __restrict__ div1, const double* __restrict__ div2, double* result) {
    if(threadIdx.x == 0) {
        *result = *div1 / *div2;
    }

}

void matrix_vector(double* matrix, double* vector, double* sol, int size) {
    for(int i = 0; i < size; i++) {
        sol[i] = 0;
        for(int j = 0; j < size; j++) {
            sol[i] += matrix[i*size + j] * vector[j];
        }
    }
}

template<int gridSize, int blockSize>
void axpby(double* __restrict__ alpha, const double * __restrict__ x, double * __restrict__ y, int size, hipStream_t stream)
{
    axpby_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(alpha, x, y, size);
}

template<int gridSize, int blockSize>
void _minus_axpby(double* __restrict__ alpha, const double * __restrict__ x, double * __restrict__ y, int size, hipStream_t stream)
{
    _minus_axpby_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(alpha, x, y, size);
}


template<int gridSize, int blockSize>
void xpby(const double * __restrict__ x, double * __restrict__ y, const double* __restrict__ beta, int size, hipStream_t stream)
{
    xpby_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(x, y, beta, size);
}



void conjugate_gradients_serial(const double * A, const double * b, double * x, size_t size, int max_iters, double rel_error, long* execution_time)
{
    double alpha, beta, bb, rr, rr_new;
    double * r = new double[size];
    double * p = new double[size];
    double * Ap = new double[size];
    int num_iters;

    for(size_t i = 0; i < size; i++)
    {
        x[i] = 0.0;
        r[i] = b[i];
        p[i] = b[i];
    }

    bb = dot(b, b, size);
    rr = bb;
    auto start = std::chrono::high_resolution_clock::now();
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        gemv(1.0, A, p, 0.0, Ap, size, size);
        alpha = rr / dot(p, Ap, size);
        axpby(alpha, p, 1.0, x, size);
        axpby(-alpha, Ap, 1.0, r, size);
        rr_new = dot(r, r, size);
        beta = rr_new / rr;
        rr = rr_new;
        if(std::sqrt(rr / bb) < rel_error) { break; }
        axpby(1.0, r, beta, p, size);
    }
    auto stop = std::chrono::high_resolution_clock::now();
    *execution_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();

    delete[] r;
    delete[] p;
    delete[] Ap;

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
}

void conjugate_gradients(const double * A, const double * b, double * x, size_t size, int max_iters, double rel_error, long* execution_time) {
    auto start = std::chrono::high_resolution_clock::now();
    double* r_cuda;
    double* p_cuda;
    double* Ap_cuda;
    double *alpha;
    double *beta;
    double* bb;
    double bb_cpu;
    double* rr;
    double* rr_new;
    double* dot_product_out_array;
    double err;
    hipMalloc(&r_cuda, size*sizeof(double));
    hipMalloc(&p_cuda, size*sizeof(double));
    hipMalloc(&Ap_cuda, size*sizeof(double));
    hipMalloc(&dot_product_out_array, sizeof(double)*GRID_SIZE);
    hipMalloc(&alpha, sizeof(double));
    hipMalloc(&beta, sizeof(double));
    hipMalloc(&bb, sizeof(double));
    hipMalloc(&rr, sizeof(double));
    hipMalloc(&rr_new, sizeof(double));
    hipMemcpy(r_cuda, b, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(p_cuda, b, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemset(x,0,sizeof(double) * size);
    int niters;
    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    dot_product<GRID_SIZE, BLOCK_SIZE>(b, b, dot_product_out_array, (int) size, bb, stream1);
    hipMemcpy(&bb_cpu, bb, sizeof(double), hipMemcpyDeviceToHost);
    err = bb_cpu;
    hipMemcpy(rr, bb, sizeof(double), hipMemcpyDeviceToDevice);

    for(niters = 1; niters < max_iters; niters++) {
        matrix_vector_mult<GRID_SIZE, BLOCK_SIZE>(A, p_cuda, Ap_cuda, (int)size, stream1);
        check_cuda("error");
        dot_product<GRID_SIZE, BLOCK_SIZE>(p_cuda, Ap_cuda, dot_product_out_array,(int)size, alpha, stream1);
        divide<<<1,1, 0, stream1>>>(rr,alpha, alpha);
        axpby<GRID_SIZE, BLOCK_SIZE>(alpha, p_cuda, x, (int)size, stream1);
        _minus_axpby<GRID_SIZE, BLOCK_SIZE>(alpha, Ap_cuda, r_cuda, (int) size, stream1);
        dot_product<GRID_SIZE, BLOCK_SIZE>(r_cuda, r_cuda, dot_product_out_array, (int)size, rr_new, stream1);
        divide<<<1, 1, 0, stream1>>>(rr_new, rr, beta);
        hipMemcpy(rr, rr_new, sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(&err, rr, sizeof(double), hipMemcpyDeviceToHost);
        if(std::sqrt(err / bb_cpu) < rel_error) { break; }
        xpby<GRID_SIZE, BLOCK_SIZE>(r_cuda, p_cuda, beta,  (int)size, stream1);
    }
    if(niters < max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", niters, std::sqrt(err / bb_cpu));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(err / bb_cpu));
    }
    hipFree(r_cuda);
    hipFree(p_cuda);
    hipFree(Ap_cuda);
    hipFree(dot_product_out_array);
    hipFree(alpha);
    hipFree(beta);
    hipFree(bb);
    hipFree(rr);
    hipFree(rr_new);
    auto stop = std::chrono::high_resolution_clock::now();
    *execution_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();

}

void print_sol(double* sol) {
    for(int i = 0; i < 5; i++) {
        std::cout << sol[i] << std::endl;
    }
}

void print_sol_cuda(double* sol) {
    double* tmp = new double[5];
    hipMemcpy(tmp, sol, 5*sizeof(double), hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; i++) {
        std::cout << tmp[i] << std::endl;
    }
}



int main(int argc, char ** argv) {

    int size = 5000;
    int max_iters = 5000;
    double rel_error = 1e-9;
    int serial_trials = 0;
    int parallel_trials = 1;
    if(argc > 1) size = atoi(argv[1]);
    if(argc > 2) max_iters = atoi(argv[2]);
    if(argc > 3) rel_error = atof(argv[3]);
    if(argc > 4) serial_trials = atoi(argv[4]);
    if(argc > 5) parallel_trials = atoi(argv[5]);

    printf("Command line arguments:\n");
    printf("  matrix_size: %d\n", size);
    printf("  max_iters:         %d\n", max_iters);
    printf("  rel_error:         %e\n", rel_error);
    printf("  serial trials number:         %d\n", serial_trials);
    printf("  parallel trials number:         %d\n", parallel_trials);
    printf("\n");

    long serial_execution_time = 0;
    long parallel_execution_time = 0;

    int* size_cuda;
    int* max_iters_cuda;
    double* tol_cuda;
    double* matrix;
    double* matrix_cuda;
    double* rhs;
    double* rhs_cuda;
    double* r_cuda;
    double* p_cuda;
    double* Ap_cuda;
    generate_matrix(size, &matrix);
    generate_rhs(size, 1.0, &rhs);
    auto* sol = new double[size];
    double* sol_cuda;

    for(int i = 0; i < size; i++) {
        sol[i] = 1.0;
    }

    hipMalloc(&matrix_cuda, size*size*sizeof(double));
    hipMalloc(&rhs_cuda, size*sizeof(double));
    hipMalloc(&sol_cuda, size*sizeof(double));
    hipMalloc(&max_iters_cuda, sizeof(int));
    hipMalloc(&size_cuda, sizeof(int));
    hipMalloc(&tol_cuda, sizeof(double));
    hipMalloc(&r_cuda, size*sizeof(double));
    hipMalloc(&p_cuda, size*sizeof(double));
    hipMalloc(&Ap_cuda, size*sizeof(double));
    hipMemcpy(matrix_cuda, matrix, size*size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rhs_cuda, rhs, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(sol_cuda, sol, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(max_iters_cuda, &max_iters, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(size_cuda, &size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tol_cuda, &rel_error, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(r_cuda, rhs, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(p_cuda, rhs, size*sizeof(double), hipMemcpyHostToDevice);


    for(int i = 0; i < serial_trials; i++) {
        long tmp;
        conjugate_gradients_serial(matrix, rhs, sol, size, max_iters, rel_error, &tmp);
        serial_execution_time += tmp;

    }
    for(int i = 0; i < parallel_trials; i++) {
        long tmp;
        conjugate_gradients(matrix_cuda, rhs_cuda, sol_cuda, size, max_iters, rel_error, &tmp);
        parallel_execution_time += tmp;
    }


    print_sol(sol);
    print_sol_cuda(sol_cuda);

    std::cout << "check" << std::endl;
    check_cuda("error");
    std::cout << "Serial average execution time: " << (double)serial_execution_time/serial_trials << std::endl;
    std::cout << "Parallel average execution time: " << (double)parallel_execution_time/parallel_trials << std::endl;
    std::cout << "Speedup: " << (double)((double)serial_execution_time/serial_trials)/((double)parallel_execution_time/parallel_trials) << std::endl;
    printf("Finished successfully\n");


}
