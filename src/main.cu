
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#define GRID_SIZE 200
#define BLOCK_SIZE 1024


void check_cuda(const std::string& msg) {
    hipDeviceSynchronize();
    hipError_t err;
    err = hipGetLastError();
    if(err != hipSuccess) {
        std::cout << "cuda error: " << msg << std::endl;
        std::cout << "description: " << err << std::endl;
    }
}

double dot(const double * x, const double * y, size_t size)
{
    double result = 0.0;
    for(size_t i = 0; i < size; i++)
    {
        result += x[i] * y[i];
    }
    return result;
}



void axpby(double alpha, const double * x, double beta, double * y, size_t size)
{
    // y = alpha * x + beta * y

    for(size_t i = 0; i < size; i++)
    {
        y[i] = alpha * x[i] + beta * y[i];
    }
}



void gemv(double alpha, const double * A, const double * x, double beta, double * y, size_t num_rows, size_t num_cols)
{
    // y = alpha * A * x + beta * y;

    for(size_t r = 0; r < num_rows; r++)
    {
        double y_val = 0.0;
        for(size_t c = 0; c < num_cols; c++)
        {
            y_val += alpha * A[r * num_cols + c] * x[c];
        }
        y[r] = beta * y[r] + y_val;
    }
}

void generate_matrix(size_t n, double** matrix_out) {
    auto* matrix = new double[n * n];
    for(size_t i = 0; i < n * n; i++) {
        matrix[i] = 0.0;
    }
    for(size_t i = 0; i < n; i++) {
        matrix[i*n + i] = 2.0;
        if(i != n-1) {
            matrix[(i+1)*n + i] = -1;
            matrix[i*n + (i+1)] = -1;
        }
    }
    *matrix_out = matrix;
}

void generate_rhs(size_t n, double value, double** rhs_out) {
    auto* rhs = new double[n];
    for(size_t i = 0; i < n; i++) {
        rhs[i] = value;
    }
    *rhs_out = rhs;
}

__device__ void warpReduce(volatile double* sdata, int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}


template<int blockSize>
__device__ void row_column_mult(const double* A, unsigned int row, int size, const double* p, double* Ap) {
    __shared__ double sArr[blockSize];
    __shared__ double partial;
    int iter_n = 0;
    if(threadIdx.x == 0) {
        partial = 0.0;
    }

    for(unsigned int i = threadIdx.x; iter_n < size; i+=2*blockSize) {
        sArr[threadIdx.x] = ((i<size)?A[row*size + i]*p[i]:0.0) + ((i + blockSize<size)?A[row*size + i + blockSize]*p[i + blockSize]:0.0);
        for (unsigned int stride = blockSize/2; stride > 32;
             stride = stride>>1)
        {

            __syncthreads();
            if (threadIdx.x < stride)
                sArr[threadIdx.x] += sArr[threadIdx.x+stride];
        }
        __syncthreads();
        if(threadIdx.x < 32) {
            warpReduce(sArr, threadIdx.x);
        }
        iter_n += 2*blockSize;
        __syncthreads();
        if(threadIdx.x == 0) {
            partial += sArr[0];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) {
        Ap[row] = partial;
    }

}

template<int gridSize, int blockSize>
__global__ void matrix_vector_kernel(const double* A, double* p, double* Ap, int size) {
    for(unsigned int i = blockIdx.x; i < size; i+=gridSize) {
        row_column_mult<blockSize>(A,i,size,p,Ap);
    }

}

template<int gridSize, int blockSize>
void matrix_vector_mult(const double* A, double* p, double* Ap, int size, hipStream_t stream) {
    matrix_vector_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(A, p, Ap, size);
}


template<int blockSize>
__global__ void sumArray(const double* array, int size, double* result) {
    __shared__ double sArr[blockSize];
    __shared__ double partial;
    int iter_n = 0;
    if(threadIdx.x == 0) {
        partial = 0;
    }
    sArr[threadIdx.x] = 0.0;
    for(unsigned int i = threadIdx.x; iter_n < size; i+=2*blockSize) {
        sArr[threadIdx.x] = ((i<size)?array[i]:0.0) + ((i + blockSize < size)?array[i + blockSize]:0.0);
        for (unsigned int stride = blockSize/2; stride > 32;
             stride = stride>>1)
        {

            __syncthreads();
            if (threadIdx.x < stride)
                sArr[threadIdx.x] += sArr[threadIdx.x+stride];
        }
        __syncthreads();
        if(threadIdx.x < 32) {
            warpReduce(sArr, threadIdx.x);
        }

        iter_n += 2*blockSize;
        __syncthreads();
        if(threadIdx.x == 0) {
            partial += sArr[0];
        }
        __syncthreads();

    }
    if(threadIdx.x == 0) {
        *result = partial;
    }
}





template<int gridSize, int blockSize>
__global__ void dot_product_kernel(const double* x, const double* y, double* outArray, int size) {
    __shared__ double sArr[blockSize];
    if(threadIdx.x == 0) {
        outArray[blockIdx.x] = 0.0;
    }
    for(unsigned int i = blockIdx.x; blockSize*i < size; i+=gridSize) {
        sArr[threadIdx.x] = ((i*2*blockSize + threadIdx.x<size)?x[i*2*blockSize + threadIdx.x]*y[i*2*blockSize + threadIdx.x]:0.0) + ((i*blockSize*2 + threadIdx.x + blockSize<size)?x[i*blockSize*2 + threadIdx.x + blockSize]*y[i*blockSize*2 + threadIdx.x + blockSize]:0.0);
        //sArr[threadIdx.x] = (i*blockSize + threadIdx.x<size)?x[i*blockSize + threadIdx.x]*y[i*blockSize + threadIdx.x]:0.0;

        for (unsigned int stride = blockSize/2; stride > 32;
             stride = stride>>1)
        {

            __syncthreads();
            if (threadIdx.x < stride)
                sArr[threadIdx.x] += sArr[threadIdx.x+stride];
        }
        __syncthreads();
        if(threadIdx.x < 32) {
            warpReduce(sArr, threadIdx.x);
        }
        __syncthreads();
        if(threadIdx.x == 0) {
            outArray[blockIdx.x] += sArr[0];
        }
        __syncthreads();
    }
}

template<int gridSize, int blockSize>
void dot_product(const double* x, const double* y, double* outArray, int size, double* result, hipStream_t stream) {
    dot_product_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(x, y, outArray, size);
    sumArray<blockSize><<<1, blockSize>>>(outArray, gridSize, result);
}

template<int gridSize, int blockSize>
__global__ void axpby_kernel(const double* alpha, const double* x, double* y, int size) {
    int th_id = threadIdx.x + blockIdx.x * blockSize;
    if(th_id < size) {
        y[th_id] = (*alpha) * x[th_id] + y[th_id];
    }
}


template<int gridSize, int blockSize>
__global__ void _minus_axpby_kernel(const double* alpha, const double* x, double* y, int size) {
    int th_id = threadIdx.x + blockIdx.x * blockSize;
    if(th_id < size) {
        y[th_id] = -(*alpha) * x[th_id] + y[th_id];
    }
}

template<int gridSize, int blockSize>
__global__ void xpby_kernel( const double* x, double* y, const double* beta, int size) {
    int th_id = threadIdx.x + blockIdx.x * blockSize;
    if(th_id < size) {
        y[th_id] = (x[th_id] + (*beta) * y[th_id]);
    }
}



__global__ void divide(double* div1, double* div2, double* result) {
    if(threadIdx.x == 0) {
        *result = *div1 / *div2;
    }

}

void matrix_vector(double* matrix, double* vector, double* sol, int size) {
    for(int i = 0; i < size; i++) {
        sol[i] = 0;
        for(int j = 0; j < size; j++) {
            sol[i] += matrix[i*size + j] * vector[j];
        }
    }
}

template<int gridSize, int blockSize>
void axpby(double* alpha, const double * x, double * y, int size, hipStream_t stream)
{
    axpby_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(alpha, x, y, size);
}

template<int gridSize, int blockSize>
void _minus_axpby(double* alpha, const double * x, double * y, int size, hipStream_t stream)
{
    _minus_axpby_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(alpha, x, y, size);
}


template<int gridSize, int blockSize>
void xpby(const double * x, double * y, const double* beta, int size, hipStream_t stream)
{
    xpby_kernel<gridSize, blockSize><<<gridSize, blockSize, 0, stream>>>(x, y, beta, size);
}



void conjugate_gradients_serial(const double * A, const double * b, double * x, size_t size, int max_iters, double rel_error, long* execution_time)
{
    double alpha, beta, bb, rr, rr_new;
    double * r = new double[size];
    double * p = new double[size];
    double * Ap = new double[size];
    int num_iters;

    for(size_t i = 0; i < size; i++)
    {
        x[i] = 0.0;
        r[i] = b[i];
        p[i] = b[i];
    }

    bb = dot(b, b, size);
    rr = bb;
    auto start = std::chrono::high_resolution_clock::now();
    for(num_iters = 1; num_iters <= max_iters; num_iters++)
    {
        gemv(1.0, A, p, 0.0, Ap, size, size);
        alpha = rr / dot(p, Ap, size);
        axpby(alpha, p, 1.0, x, size);
        axpby(-alpha, Ap, 1.0, r, size);
        rr_new = dot(r, r, size);
        beta = rr_new / rr;
        rr = rr_new;
        if(std::sqrt(rr / bb) < rel_error) { break; }
        axpby(1.0, r, beta, p, size);
    }
    auto stop = std::chrono::high_resolution_clock::now();
    *execution_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();

    delete[] r;
    delete[] p;
    delete[] Ap;

    if(num_iters <= max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", num_iters, std::sqrt(rr / bb));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(rr / bb));
    }
}

void conjugate_gradients(const double * A, const double * b, double * x, size_t size, int max_iters, double rel_error, long* execution_time) {
    double* r_cuda;
    double* p_cuda;
    double* Ap_cuda;
    double *alpha;
    double *beta;
    double* bb;
    double bb_cpu;
    double* rr;
    double* rr_new;
    double* dot_product_out_array;
    double err;
    hipMalloc(&r_cuda, size*sizeof(double));
    hipMalloc(&p_cuda, size*sizeof(double));
    hipMalloc(&Ap_cuda, size*sizeof(double));
    hipMalloc(&dot_product_out_array, sizeof(double)*GRID_SIZE);
    hipMalloc(&alpha, sizeof(double));
    hipMalloc(&beta, sizeof(double));
    hipMalloc(&bb, sizeof(double));
    hipMalloc(&rr, sizeof(double));
    hipMalloc(&rr_new, sizeof(double));
    hipMemcpy(r_cuda, b, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(p_cuda, b, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemset(x,0,sizeof(double) * size);
    int niters;
    hipStream_t stream1;
    hipStream_t stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    dot_product<GRID_SIZE, BLOCK_SIZE>(b, b, dot_product_out_array, (int) size, bb, stream1);
    hipMemcpy(&bb_cpu, bb, sizeof(double), hipMemcpyDeviceToHost);
    err = bb_cpu;
    hipMemcpy(rr, bb, sizeof(double), hipMemcpyDeviceToDevice);
    auto start = std::chrono::high_resolution_clock::now();
    for(niters = 1; niters < max_iters; niters++) {
        matrix_vector_mult<GRID_SIZE, BLOCK_SIZE>(A, p_cuda, Ap_cuda, (int)size, stream1);
        dot_product<GRID_SIZE, BLOCK_SIZE>(p_cuda, Ap_cuda, dot_product_out_array,(int)size, alpha, stream1);
        divide<<<1,1, 0, stream1>>>(rr,alpha, alpha);
        axpby<GRID_SIZE, BLOCK_SIZE>(alpha, p_cuda, x, (int)size, stream1);
        _minus_axpby<GRID_SIZE, BLOCK_SIZE>(alpha, Ap_cuda, r_cuda, (int) size, stream1);
        dot_product<GRID_SIZE, BLOCK_SIZE>(r_cuda, r_cuda, dot_product_out_array, (int)size, rr_new, stream1);
        divide<<<1, 1, 0, stream1>>>(rr_new, rr, beta);
        hipMemcpy(rr, rr_new, sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(&err, rr, sizeof(double), hipMemcpyDeviceToHost);
        if(std::sqrt(err / bb_cpu) < rel_error) { break; }
        xpby<GRID_SIZE, BLOCK_SIZE>(r_cuda, p_cuda, beta,  (int)size, stream1);
    }
    auto stop = std::chrono::high_resolution_clock::now();
    if(niters < max_iters)
    {
        printf("Converged in %d iterations, relative error is %e\n", niters, std::sqrt(err / bb_cpu));
    }
    else
    {
        printf("Did not converge in %d iterations, relative error is %e\n", max_iters, std::sqrt(err / bb_cpu));
    }
    *execution_time = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    hipFree(r_cuda);
    hipFree(p_cuda);
    hipFree(Ap_cuda);
    hipFree(dot_product_out_array);
    hipFree(alpha);
    hipFree(beta);
    hipFree(bb);
    hipFree(rr);
    hipFree(rr_new);
}

void print_sol(double* sol) {
    for(int i = 0; i < 5; i++) {
        std::cout << sol[i] << std::endl;
    }
}

void print_sol_cuda(double* sol) {
    double* tmp = new double[5];
    hipMemcpy(tmp, sol, 5*sizeof(double), hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; i++) {
        std::cout << tmp[i] << std::endl;
    }
}



int main(int argc, char ** argv) {

    int size = 500;
    int max_iters = 1000;
    double rel_error = 1e-9;
    int serial_trials = 1;
    int parallel_trials = 1;
    if(argc > 1) size = atoi(argv[1]);
    if(argc > 2) max_iters = atoi(argv[2]);
    if(argc > 3) rel_error = atof(argv[3]);
    if(argc > 4) serial_trials = atoi(argv[4]);
    if(argc > 5) parallel_trials = atoi(argv[5]);

    printf("Command line arguments:\n");
    printf("  matrix_size: %d\n", size);
    printf("  max_iters:         %d\n", max_iters);
    printf("  rel_error:         %e\n", rel_error);
    printf("  serial trials number:         %d\n", serial_trials);
    printf("  parallel trials number:         %d\n", parallel_trials);
    printf("\n");

    long serial_execution_time = 0;
    long parallel_execution_time = 0;

    int* size_cuda;
    int* max_iters_cuda;
    double* tol_cuda;
    double* matrix;
    double* matrix_cuda;
    double* rhs;
    double* rhs_cuda;
    double* r_cuda;
    double* p_cuda;
    double* Ap_cuda;
    generate_matrix(size, &matrix);
    generate_rhs(size, 2.0, &rhs);
    auto* sol = new double[size];
    double* sol_cuda;

    for(int i = 0; i < size; i++) {
        sol[i] = 1.0;
    }

    hipMalloc(&matrix_cuda, size*size*sizeof(double));
    hipMalloc(&rhs_cuda, size*sizeof(double));
    hipMalloc(&sol_cuda, size*sizeof(double));
    hipMalloc(&max_iters_cuda, sizeof(int));
    hipMalloc(&size_cuda, sizeof(int));
    hipMalloc(&tol_cuda, sizeof(double));
    hipMalloc(&r_cuda, size*sizeof(double));
    hipMalloc(&p_cuda, size*sizeof(double));
    hipMalloc(&Ap_cuda, size*sizeof(double));
    hipMemcpy(matrix_cuda, matrix, size*size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rhs_cuda, rhs, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(sol_cuda, sol, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(max_iters_cuda, &max_iters, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(size_cuda, &size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(tol_cuda, &rel_error, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(r_cuda, rhs, size*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(p_cuda, rhs, size*sizeof(double), hipMemcpyHostToDevice);


    for(int i = 0; i < serial_trials; i++) {
        long tmp;
        conjugate_gradients_serial(matrix, rhs, sol, size, max_iters, rel_error, &tmp);
        serial_execution_time += tmp;

    }
    for(int i = 0; i < parallel_trials; i++) {
        long tmp;
        conjugate_gradients(matrix_cuda, rhs_cuda, sol_cuda, size, max_iters, rel_error, &tmp);
        parallel_execution_time += tmp;
    }


    print_sol(sol);
    print_sol_cuda(sol_cuda);

    std::cout << "check" << std::endl;
    check_cuda("error");
    std::cout << "Serial average execution time: " << (double)serial_execution_time/serial_trials << std::endl;
    std::cout << "Parallel average execution time: " << (double)parallel_execution_time/parallel_trials << std::endl;
    std::cout << "Speedup: " << (double)((double)serial_execution_time/serial_trials)/((double)parallel_execution_time/parallel_trials) << std::endl;
    printf("Finished successfully\n");


}
